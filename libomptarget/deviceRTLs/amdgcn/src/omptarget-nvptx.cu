#include "hip/hip_runtime.h"
//===--- omptarget-nvptx.cu - NVPTX OpenMP GPU initialization ---- CUDA -*-===//
//
//                     The LLVM Compiler Infrastructure
//
// This file is dual licensed under the MIT and the University of Illinois Open
// Source Licenses. See LICENSE.txt for details.
//
//===----------------------------------------------------------------------===//
//
// This file contains the initialization code for the GPU
//
//===----------------------------------------------------------------------===//

#include "omptarget-nvptx.h"

////////////////////////////////////////////////////////////////////////////////
// init entry points
////////////////////////////////////////////////////////////////////////////////

#ifdef GPUCC_AMDGCN
INLINE unsigned smid() {
  // return __smid();
  // For amdgcn, use a virtual smid based on global thread number
  unsigned id = ((blockIdx.x*blockDim.x) + threadIdx.x) / warpSize;
  PRINT(LD_IO, "smid() returns value %d with MAX_SM %d\n",id,MAX_SM);
  return id;
}
#else
INLINE unsigned smid() {
  unsigned id;
  asm("mov.u32 %0, %%smid;" : "=r"(id));
  return id;
}

INLINE unsigned n_sm() {
  unsigned n_sm;
  asm("mov.u32 %0, %%nsmid;" : "=r"(n_sm));
  return n_sm;
}
#endif

EXTERN void __kmpc_kernel_init(int ThreadLimit,
                               int16_t RequiresOMPRuntime) {
  PRINT(LD_IO, "call to __kmpc_kernel_init with version %f, threadlimit %d\n",
      OMPTARGET_NVPTX_VERSION, ThreadLimit);

  if (!RequiresOMPRuntime) {
    PRINT0(LD_IO, "OMP runtime not required\n");
    // If OMP runtime is not required don't initialize OMP state.
    setExecutionParameters(Generic, RuntimeUninitialized);
    return;
  }
  setExecutionParameters(Generic, RuntimeInitialized);

  int threadIdInBlock = GetThreadIdInBlock();
  ASSERT0(LT_FUSSY, threadIdInBlock == GetMasterThreadID(),
          "__kmpc_kernel_init() must be called by team master warp only!");
  PRINT0(LD_IO, "call to __kmpc_kernel_init for master\n");

  // Get a state object from the queue.
  int slot = smid() % MAX_SM;
  omptarget_nvptx_threadPrivateContext = omptarget_nvptx_device_State[slot].Dequeue();

  // init thread private
  int threadId = GetLogicalThreadIdInBlock();
  omptarget_nvptx_threadPrivateContext->InitThreadPrivateContext(
      threadId);

  // init team context
  omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
  currTeamDescr.InitTeamDescr();
  // this thread will start execution... has to update its task ICV
  // to point to the level zero task ICV. That ICV was init in
  // InitTeamDescr()
  omptarget_nvptx_threadPrivateContext->SetTopLevelTaskDescr(
      threadId, currTeamDescr.LevelZeroTaskDescr());

  // set number of threads and thread limit in team to started value
  omptarget_nvptx_TaskDescr *currTaskDescr =
      omptarget_nvptx_threadPrivateContext->GetTopLevelTaskDescr(
          threadId);
  currTaskDescr->NThreads() = GetNumberOfWorkersInTeam();
  currTaskDescr->ThreadLimit() = ThreadLimit;
}

EXTERN void __kmpc_kernel_deinit(int16_t IsOMPRuntimeInitialized) {
  PRINT(LD_IO, "call to __kmpc_kernel_deinit, IsOMPRuntimeInitialized %d\n",
      IsOMPRuntimeInitialized);

  if (IsOMPRuntimeInitialized) {
    // Enqueue omp state object for use by another team.
    int slot = smid() % MAX_SM;
    omptarget_nvptx_device_State[slot].Enqueue(omptarget_nvptx_threadPrivateContext);
  }
  // Done with work.  Kill the workers.
  omptarget_nvptx_workFn = 0;
}

EXTERN void __kmpc_spmd_kernel_init(int ThreadLimit,
                                    int16_t RequiresOMPRuntime,
                                    int16_t RequiresDataSharing) {
  PRINT(LD_IO, "call to __kmpc_spmd_kernel_init with version %f, threadlimit %d\n",
      OMPTARGET_NVPTX_VERSION, ThreadLimit);

  if (!RequiresOMPRuntime) {
    PRINT0(LD_IO, "OMP runtime not required\n");
    // If OMP runtime is not required don't initialize OMP state.
    setExecutionParameters(Spmd, RuntimeUninitialized);
    return;
  }
  setExecutionParameters(Spmd, RuntimeInitialized);

  //
  // Team Context Initialization.
  //
  // In SPMD mode there is no master thread so use any cuda thread for team
  // context initialization.
  int threadId = GetThreadIdInBlock();
  if (threadId == 0) {
    // Get a state object from the queue.
    int slot = smid() % MAX_SM;
    omptarget_nvptx_threadPrivateContext = omptarget_nvptx_device_State[slot].Dequeue();

    omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
    omptarget_nvptx_WorkDescr &workDescr = getMyWorkDescriptor();
    // init team context
    currTeamDescr.InitTeamDescr();
    // init counters (copy start to init)
    workDescr.CounterGroup().Reset();
  }
  __syncthreads();

  omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
  omptarget_nvptx_WorkDescr &workDescr = getMyWorkDescriptor();

  //
  // Initialize task descr for each thread.
  //
  omptarget_nvptx_TaskDescr *newTaskDescr =
      omptarget_nvptx_threadPrivateContext->Level1TaskDescr(threadId);
  ASSERT0(LT_FUSSY, newTaskDescr, "expected a task descr");
  newTaskDescr->InitLevelOneTaskDescr(
    ThreadLimit, currTeamDescr.LevelZeroTaskDescr());
  // install new top descriptor
  omptarget_nvptx_threadPrivateContext->SetTopLevelTaskDescr(threadId,
                                                             newTaskDescr);

  // init thread private from init value
  workDescr.CounterGroup().Init(
      omptarget_nvptx_threadPrivateContext->Priv(threadId));
  PRINT(LD_PAR, "thread will execute parallel region with id %d in a team of "
                "%d threads\n",
                newTaskDescr->ThreadId(), newTaskDescr->NThreads());

  if (RequiresDataSharing && threadId % WARPSIZE == 0) {
    // Warp master innitializes data sharing environment.
    unsigned WID = threadId >> DS_Max_Worker_Warp_Size_Log2;
    __kmpc_data_sharing_slot *RootS = currTeamDescr.RootS(WID);
    DataSharingState.SlotPtr[WID] = RootS;
    DataSharingState.StackPtr[WID] = (void*)&RootS->Data[0];
  }
}

EXTERN void __kmpc_spmd_kernel_deinit() {
  PRINT0(LD_IO, "call to __kmpc_spmd_kernel_deinit\n");

  // We're not going to pop the task descr stack of each thread since
  // there are no more parallel regions in SPMD mode.
  __syncthreads();
  int threadId = GetThreadIdInBlock();
  if (threadId == 0) {
    // Enqueue omp state object for use by another team.
    int slot = smid() % MAX_SM;
    omptarget_nvptx_device_State[slot].Enqueue(omptarget_nvptx_threadPrivateContext);
  }
}
